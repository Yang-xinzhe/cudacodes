#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "naive_2d.cuh"

__global__ void naive_xgemm_2d_kernel(float* __restrict__ A_gpu, float* __restrict__ B_gpu, float* __restrict__ C_gpu, int M, int N, int K) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if(row < M && col < N) {
        float sum = 0.0f;
        for(int k = 0 ; k < K ; ++k) {
            sum += A_gpu[row * K + k] * B_gpu[k * N + col];
        }
        C_gpu[row * N + col] = sum;
    }
}

float run_naive_xgemm_2d_kernel(float* __restrict__ A_gpu, float* __restrict__ B_gpu, float* __restrict__ C_gpu, int M, int N, int K){
    const int BLOCK_DIM = 16;
    dim3 block_size(BLOCK_DIM, BLOCK_DIM); // 16x16=256 threads
    dim3 grid_size(((N + block_size.x - 1) / block_size.x), (M + block_size.y - 1) / block_size.y);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float ms = 0.0f;

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(naive_xgemm_2d_kernel), hipFuncCachePreferL1);
    hipDeviceSynchronize();
    hipEventRecord(start);
    naive_xgemm_2d_kernel<<<grid_size, block_size>>>(A_gpu, B_gpu, C_gpu, M, N, K);
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipDeviceSynchronize();
    hipEventElapsedTime(&ms, start, end);
    printf(">> naive_xgemm_2d_kernel execute time: %.3f ms\n", ms);
    return ms;
}