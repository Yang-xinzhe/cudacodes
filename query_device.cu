#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int dev_count;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&dev_count);
    hipGetDeviceProperties(&prop, 0);

    printf(">> CUDA enabled devices in the system: %d\n", dev_count);
    printf(">> Compute capability: %d.%d\n", prop.major, prop.minor);

    printf(">> Max grid size: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf(">> Max block size: %d\n", prop.maxThreadsPerBlock);

    printf(">> Number of SMs: %d\n", prop.multiProcessorCount);
    printf(">> Clock Rate of the SMs (in kHz): %d\n", prop.clockRate);

    printf(">> Max threads dimension: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf(">> Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor); // 1536 / 256 = 6 blocks per SM

    printf(">> Registers available per block: %d\n", prop.regsPerBlock); // 65536
    printf(">> Registers available per SM: %d\n", prop.regsPerMultiprocessor); // 65536

    printf(">> Warp size (threads per warp): %d\n", prop.warpSize);
    printf(">> Shared memory size per block: %zd bytes\n", prop.sharedMemPerBlock); // 48 KB
    printf(">> Shared memory size per SM: %zd bytes\n", prop.sharedMemPerMultiprocessor); // 100 KB

    // CUDA Core → Registers → Shared Memory → L1 Cache → L2 Cache → Global Memory (VRAM)
    printf(">> L2 Cache size: %d bytes\n", prop.l2CacheSize); // 35 MB

    printf(">> Memory bus width: %d bits\n", prop.memoryBusWidth);
    printf(">> Memory clock rate: %d kHz\n", prop.memoryClockRate);

    int cudaCores = prop.multiProcessorCount * 128;
    float clock_GHz = prop.clockRate / 1e6;
    float gflops = cudaCores * clock_GHz * 2; // Fused Multiply-Add， two float calculate

    printf(">> Theoretical Max GFLOPS: %.2f\n", gflops);

    float memoryBandwidth = (2 * prop.memoryClockRate * prop.memoryBusWidth) / (8.0 * 1e6);
    printf(">> Maximum Memory Bandwidth: %.2f GB/s\n", memoryBandwidth);
}